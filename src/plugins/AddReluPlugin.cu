#include "hip/hip_runtime.h"
#include "AddReluPlugin.h"

using namespace nvinfer1;

PluginFieldCollection    AddReluPluginCreator::fc_ {};
std::vector<PluginField> AddReluPluginCreator::attr_;

template <int TPB, int VPT>
__global__ void add_relu(
    const int ld, const float* input, float* output, const float* beta)
{
    const int idx = ld * blockIdx.x + threadIdx.x * VPT;
    float in_local[VPT];
    float beta_local[VPT];
    float out_local[VPT];
    copy<sizeof(float) * VPT>(&input[idx], in_local);
    copy<sizeof(float) * VPT>(&beta[threadIdx.x * VPT], beta_local);

#pragma unroll
    for (int it = 0; it < VPT; it++)
    {
        out_local[it] = max(in_local[it] + beta_local[it], 0.0f);
    }

    copy<sizeof(float) * VPT>(out_local, &output[idx]);
}

template <int TPB, int VPT>
__global__ void add_relu_half(
    const int ld, const half* input, half* output, const half* beta)
{
    const int idx = ld * blockIdx.x + threadIdx.x * VPT;
    half in_local[VPT];
    half beta_local[VPT];
    half out_local[VPT];
    copy<sizeof(half) * VPT>(&input[idx], in_local);
    copy<sizeof(half) * VPT>(&beta[threadIdx.x * VPT], beta_local);

#pragma unroll
    for (int it = 0; it < VPT; it++)
    {
        out_local[it] = __hmax(in_local[it] + beta_local[it], (half)0.0f);
    }

    copy<sizeof(half) * VPT>(out_local, &output[idx]);
}

int32_t AddReluPlugin::enqueue(const PluginTensorDesc *inputDesc, const PluginTensorDesc *outputDesc, const void *const *inputs, void *const *outputs, void *workspace, hipStream_t stream) noexcept
{
    WHERE_AM_I();
    int nBlock = 1;
    for(int i = 0; i < inputDesc[0].dims.nbDims - 1; ++i)
        nBlock *= inputDesc[0].dims.d[i];
    if (inputDesc[0].type == DataType::kFLOAT)
    {
        constexpr int VPT = 4;
        constexpr int TPB = 768;
        add_relu<TPB, VPT><<<nBlock, TPB, 0, stream>>>(3072, (float *)inputs[0], (float *)outputs[0], (float *)inputs[1]);
    }
    else if (inputDesc[0].type == DataType::kHALF)
    {
        constexpr int VPT = 4;
        constexpr int TPB = 768;
        add_relu_half<TPB, VPT><<<nBlock, TPB, 0, stream>>>(3072, (half *)inputs[0], (half *)outputs[0], (half *)inputs[1]);
    }
    return 0;
}

REGISTER_TENSORRT_PLUGIN(AddReluPluginCreator);
