#include "hip/hip_runtime.h"
#include "AddResidualLayerNormPlugin.h"

using namespace nvinfer1;

PluginFieldCollection    AddResidualLayerNormPluginCreator::fc_ {};
std::vector<PluginField> AddResidualLayerNormPluginCreator::attr_;

template <int TPB, int VPT>
__global__ void add_residual_ln_vec(
    const int ld, const half* residual, const half* input, half* output, const half* beta, const half* gamma)
{
    const int idx = ld * blockIdx.x + threadIdx.x * VPT;
    // 4 * 1024 * 4 * 2 Bytes = 16KB per block
    half in_local[VPT];
    half residual_local[VPT];
    half beta_local[VPT];
    half gamma_local[VPT];
    copy<sizeof(half) * VPT>(&input[idx], in_local);
    copy<sizeof(half) * VPT>(&residual[idx], residual_local);
    float local = 0.f;
    float local2 = 0.f;

    const float rld = float(1) / float(ld);
#pragma unroll
    for (int it = 0; it < VPT; it++)
    {
        in_local[it] += residual_local[it];
        const float tmp = rld * __half2float(in_local[it]);
        local += tmp;
        local2 += tmp * __half2float(in_local[it]);
    }

    copy<sizeof(half) * VPT>(&gamma[threadIdx.x * VPT], gamma_local);
    copy<sizeof(half) * VPT>(&beta[threadIdx.x * VPT], beta_local);

    using BlockReduce = hipcub::BlockReduce<kvp<float>, TPB>;
    __shared__ typename BlockReduce::TempStorage temp_storage;
    __shared__ float mu;     // mean
    __shared__ float rsigma; // 1 / std.dev.

    const auto sumKV = BlockReduce(temp_storage).Reduce(kvp<float>(local, local2), hipcub::Sum());

    if (threadIdx.x == 0)
    {
        mu = sumKV.key;
        rsigma = rsqrt(sumKV.value - mu * mu );
    }
    __syncthreads();
    ///*
#pragma unroll
    for (int it = 0; it < VPT; it++)
    {
        in_local[it] = gamma_local[it] * (in_local[it] - __float2half(mu)) * __float2half(rsigma) + beta_local[it];
    }
    /* */

    copy<sizeof(half) * VPT>(in_local, &output[idx]);
}

template <int TPB, int VPT>
__global__ void add_residual_ln_vec(
    const int ld, const float* residual, const float* input, float* output, const float* beta, const float* gamma)
{
    const int idx = ld * blockIdx.x + threadIdx.x * VPT;
    // 4 * 1024 * 4 * 2 Bytes = 16KB per block
    float in_local[VPT];
    float residual_local[VPT];
    float beta_local[VPT];
    float gamma_local[VPT];
    float sum_local[VPT];
    copy<sizeof(float) * VPT>(&input[idx], in_local);
    copy<sizeof(float) * VPT>(&residual[idx], residual_local);
    float local = 0.f;
    float local2 = 0.f;

#pragma unroll
    for (int it = 0; it < VPT; it++)
    {
        sum_local[it] = in_local[it]  + residual_local[it];
    }

    const float rld = float(1) / float(ld);
#pragma unroll
    for (int it = 0; it < VPT; it++)
    {
        const float tmp = rld * sum_local[it];
        local += tmp;
        local2 += tmp * sum_local[it];
    }

    copy<sizeof(float) * VPT>(&gamma[threadIdx.x * VPT], gamma_local);
    copy<sizeof(float) * VPT>(&beta[threadIdx.x * VPT], beta_local);

    using BlockReduce = hipcub::BlockReduce<kvp<float>, TPB>;
    __shared__ typename BlockReduce::TempStorage temp_storage;
    __shared__ float mu;     // mean
    __shared__ float rsigma; // 1 / std.dev.

    const auto sumKV = BlockReduce(temp_storage).Reduce(kvp<float>(local, local2), hipcub::Sum());

    if (threadIdx.x == 0)
    {
        mu = sumKV.key;
        rsigma = rsqrt(sumKV.value - mu * mu );
    }
    __syncthreads();
    ///*
#pragma unroll
    for (int it = 0; it < VPT; it++)
    {
        in_local[it] = gamma_local[it] * (sum_local[it] - mu) * rsigma + beta_local[it];
    }
    /* */

    copy<sizeof(float) * VPT>(in_local, &output[idx]);
}

int32_t AddResidualLayerNormPlugin::enqueue(const PluginTensorDesc *inputDesc, const PluginTensorDesc *outputDesc, const void *const *inputs, void *const *outputs, void *workspace, hipStream_t stream) noexcept
{
    WHERE_AM_I();
    int nBlock = 1;
    for(int i = 0; i < inputDesc[0].dims.nbDims - 1; ++i)
        nBlock *= inputDesc[0].dims.d[i];
    if (inputDesc[0].type == DataType::kHALF)
    {
        constexpr int VPT = 4;
        constexpr int TPB = 768 / VPT;
        add_residual_ln_vec<TPB, VPT><<<nBlock, TPB, 0, stream>>>(768, (half *)inputs[0], (half *)inputs[1], (half *)outputs[0], (half *)inputs[3], (half *)inputs[2]);
    }
    return 0;
}

REGISTER_TENSORRT_PLUGIN(AddResidualLayerNormPluginCreator);
